
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK_ERROR(err) if (err != hipSuccess) { \
  printf("CUDA Error: %s\n", hipGetErrorString(err)); \
  return -1; \
}

__global__ void myKernel() {
    printf("Hello, World from GPU!\n");
}

int main() {
    myKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();  // Check kernel launch
    CUDA_CHECK_ERROR(err);

    err = hipDeviceSynchronize();  // Check runtime errors
    CUDA_CHECK_ERROR(err);
    return 0;
}
